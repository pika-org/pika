#include "hip/hip_runtime.h"
//  Copyright (c) 2021 ETH Zurich
//
//  SPDX-License-Identifier: BSL-1.0
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <pika/cuda.hpp>
#include <pika/execution.hpp>
#include <pika/init.hpp>
#include <pika/testing.hpp>

#include "algorithm_test_utils.hpp"

#include <atomic>
#include <cstddef>
#include <utility>

__global__ void dummy_kernel() {}

struct dummy
{
    static std::atomic<std::size_t> host_void_calls;
    static std::atomic<std::size_t> stream_void_calls;
    static std::atomic<std::size_t> cublas_void_calls;
    static std::atomic<std::size_t> cusolver_void_calls;
    static std::atomic<std::size_t> host_int_calls;
    static std::atomic<std::size_t> stream_int_calls;
    static std::atomic<std::size_t> cublas_int_calls;
    static std::atomic<std::size_t> cusolver_int_calls;
    static std::atomic<std::size_t> host_double_calls;
    static std::atomic<std::size_t> stream_double_calls;
    static std::atomic<std::size_t> cublas_double_calls;
    static std::atomic<std::size_t> cusolver_double_calls;

    static void reset_counts()
    {
        host_void_calls = 0;
        stream_void_calls = 0;
        cublas_void_calls = 0;
        cusolver_void_calls = 0;
        host_int_calls = 0;
        stream_int_calls = 0;
        cublas_int_calls = 0;
        cusolver_int_calls = 0;
        host_double_calls = 0;
        stream_double_calls = 0;
        cublas_double_calls = 0;
        cusolver_double_calls = 0;
    }

    void operator()() const
    {
        ++host_void_calls;
    }

    void operator()(whip::stream_t stream) const
    {
        ++stream_void_calls;
        dummy_kernel<<<1, 1, 0, stream>>>();
    }

    void operator()(hipblasHandle_t) const
    {
        ++cublas_void_calls;
    }

    void operator()(hipsolverHandle_t) const
    {
        ++cusolver_void_calls;
    }

    double operator()(int x) const
    {
        ++host_int_calls;
        return x + 1;
    }

    double operator()(int x, whip::stream_t stream) const
    {
        ++stream_int_calls;
        dummy_kernel<<<1, 1, 0, stream>>>();
        return x + 1;
    }

    double operator()(hipblasHandle_t, int x) const
    {
        ++cublas_int_calls;
        return x + 1;
    }

    double operator()(hipsolverHandle_t, int x) const
    {
        ++cusolver_int_calls;
        return x + 1;
    }

    int operator()(double x) const
    {
        ++host_double_calls;
        return x + 1;
    }

    int operator()(double x, whip::stream_t stream) const
    {
        ++stream_double_calls;
        dummy_kernel<<<1, 1, 0, stream>>>();
        return x + 1;
    }

    int operator()(hipblasHandle_t, double x) const
    {
        ++cublas_double_calls;
        return x + 1;
    }

    int operator()(hipsolverHandle_t, double x) const
    {
        ++cusolver_double_calls;
        return x + 1;
    }
};

std::atomic<std::size_t> dummy::host_void_calls{0};
std::atomic<std::size_t> dummy::stream_void_calls{0};
std::atomic<std::size_t> dummy::cublas_void_calls{0};
std::atomic<std::size_t> dummy::cusolver_void_calls{0};
std::atomic<std::size_t> dummy::host_int_calls{0};
std::atomic<std::size_t> dummy::stream_int_calls{0};
std::atomic<std::size_t> dummy::cublas_int_calls{0};
std::atomic<std::size_t> dummy::cusolver_int_calls{0};
std::atomic<std::size_t> dummy::host_double_calls{0};
std::atomic<std::size_t> dummy::stream_double_calls{0};
std::atomic<std::size_t> dummy::cublas_double_calls{0};
std::atomic<std::size_t> dummy::cusolver_double_calls{0};

struct dummy_stream
{
    bool& called;
    void operator()(whip::stream_t)
    {
        called = true;
    }
};

struct dummy_cublas
{
    bool& called;
    void operator()(hipblasHandle_t)
    {
        called = true;
    }
};

struct dummy_cusolver
{
    bool& called;
    void operator()(hipsolverHandle_t)
    {
        called = true;
    }
};

__global__ void increment_kernel(int* p)
{
    ++(*p);
}

struct increment
{
    int* operator()(int* p, whip::stream_t stream) const
    {
        increment_kernel<<<1, 1, 0, stream>>>(p);
        return p;
    }
};

auto non_default_constructible_params(
    custom_type_non_default_constructible& x, whip::stream_t)
{
    return std::move(x);
}
auto non_default_constructible_non_copyable_params(
    custom_type_non_default_constructible_non_copyable& x, whip::stream_t)
{
    return std::move(x);
}

int pika_main()
{
    namespace cu = ::pika::cuda::experimental;
    namespace ex = ::pika::execution::experimental;
    namespace tt = ::pika::this_thread::experimental;

    cu::cuda_pool pool{};

    cu::enable_user_polling p;

    // Only stream transform
    {
        dummy::reset_counts();
        auto s = ex::just() | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{});
        // NOTE: then_with_stream calls triggers the receiver on a plain
        // std::thread. We explicitly change the context back to an pika::thread.
        tt::sync_wait(ex::transfer(std::move(s), ex::thread_pool_scheduler{}));
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    {
        dummy::reset_counts();
        auto s = ex::just() | ex::transfer(cu::cuda_scheduler(pool)) |
            cu::then_with_stream(dummy{}) | cu::then_with_stream(dummy{}) |
            cu::then_with_stream(dummy{});
        tt::sync_wait(ex::transfer(std::move(s), ex::thread_pool_scheduler{}));
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(3));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    // Mixing stream transform with host scheduler
    {
        dummy::reset_counts();
        auto s = ex::just() | ex::transfer(cu::cuda_scheduler(pool)) |
            cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{}) |
            ex::transfer(cu::cuda_scheduler(pool)) |
            cu::then_with_stream(dummy{});
        tt::sync_wait(ex::transfer(std::move(s), ex::thread_pool_scheduler{}));
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    {
        dummy::reset_counts();
        auto s = ex::schedule(ex::thread_pool_scheduler{}) | ex::then(dummy{}) |
            ex::transfer(cu::cuda_scheduler(pool)) |
            cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        tt::sync_wait(std::move(s));
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    // Only stream transform with non-void values
    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(
                         std::move(s), ex::thread_pool_scheduler{})),
            2.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) | cu::then_with_stream(dummy{}) |
            cu::then_with_stream(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(
                         std::move(s), ex::thread_pool_scheduler{})),
            4.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
    }

    // Non-copyable or non-default-constructible types
    {
        auto s = ex::just(custom_type_non_default_constructible{42}) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(&non_default_constructible_params);
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(std::move(s),
                                       ex::thread_pool_scheduler{}))
                         .x,
            42);
    }

    {
        auto s =
            ex::just(custom_type_non_default_constructible_non_copyable{42}) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(
                &non_default_constructible_non_copyable_params);
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(std::move(s),
                                       ex::thread_pool_scheduler{}))
                         .x,
            42);
    }

    // Mixing stream transform with host scheduler with non-void values
    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{}) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(
                         std::move(s), ex::thread_pool_scheduler{})),
            4.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then(dummy{}) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(std::move(s)), 4.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
    }

    {
        dummy::reset_counts();

        auto s = ex::transfer_just(ex::thread_pool_scheduler{}, 1) |
            ex::then(dummy{}) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) | cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(std::move(s)), 5.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
    }

    // Chaining multiple stream transforms without intermediate synchronization
    {
        whip::stream_t first_stream{};
        whip::stream_t second_stream{};
        tt::sync_wait(ex::schedule(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(
                [&](whip::stream_t stream) { first_stream = stream; }) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_EQ(stream, first_stream);
            }) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_EQ(stream, first_stream);
            }) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_NEQ(stream, first_stream);
                second_stream = stream;
            }) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_EQ(stream, second_stream);
            }) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_EQ(stream, second_stream);
            }));
    }

    {
        using type = int;
        type p_h = 0;

        type* p;
        whip::malloc(&p, sizeof(type));

        auto s = ex::just(p, &p_h, sizeof(type), whip::memcpy_host_to_device) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(whip::memcpy_async) |
            ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then([p] { return p; }) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(increment{}) |
            cu::then_with_stream(increment{}) |
            cu::then_with_stream(increment{});
        tt::sync_wait(
            ex::when_all(ex::just(&p_h), std::move(s), ex::just(sizeof(type)),
                ex::just(whip::memcpy_device_to_host)) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(whip::memcpy_async) |
            ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then([&p_h] { PIKA_TEST_EQ(p_h, 3); }) |
            ex::transfer(ex::thread_pool_scheduler{}));

        whip::free(p);
    }

    // cuBLAS and cuSOLVER
    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then(dummy{}) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) |
            cu::then_with_cublas(dummy{}, HIPBLAS_POINTER_MODE_HOST) |
            cu::then_with_cusolver(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(std::move(s)), 6);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cublas_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cusolver_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cublas_int_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::cusolver_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::cublas_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cusolver_double_calls.load(), std::size_t(1));
    }

    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then(dummy{}) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) | cu::then_on_host(dummy{}) |
            cu::then_with_cublas(dummy{}, HIPBLAS_POINTER_MODE_HOST) |
            cu::then_with_cusolver(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(std::move(s)), 7.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cublas_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cusolver_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cublas_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cusolver_int_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::cublas_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::cusolver_double_calls.load(), std::size_t(0));
    }

    return pika::finalize();
}

int main(int argc, char* argv[])
{
    PIKA_TEST_EQ_MSG(pika::init(pika_main, argc, argv), 0,
        "pika main exited with non-zero status");

    return 0;
}
