#include "hip/hip_runtime.h"
//  Copyright (c) 2021 ETH Zurich
//
//  SPDX-License-Identifier: BSL-1.0
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <pika/cuda.hpp>
#include <pika/execution.hpp>
#include <pika/init.hpp>
#include <pika/testing.hpp>

#include <whip.hpp>

#include "algorithm_test_utils.hpp"

#include <atomic>
#include <cstddef>
#include <exception>
#include <functional>
#include <string>
#include <utility>

namespace cu = pika::cuda::experimental;
namespace ex = pika::execution::experimental;
namespace tt = pika::this_thread::experimental;

__global__ void dummy_kernel() {}

template <typename T>
struct const_reference_cuda_sender
{
    std::reference_wrapper<std::decay_t<T>> x;
    cu::cuda_scheduler sched;

    template <template <class...> class Tuple,
        template <class...> class Variant>
    using value_types = Variant<Tuple<std::decay_t<T> const&>>;

    template <template <class...> class Variant>
    using error_types = Variant<std::exception_ptr>;

    static constexpr bool sends_done = false;

    using completion_signatures =
        pika::execution::experimental::completion_signatures<
            pika::execution::experimental::set_value_t(std::decay_t<T>&),
            pika::execution::experimental::set_error_t(std::exception_ptr)>;

    template <typename R>
    struct operation_state
    {
        std::reference_wrapper<std::decay_t<T>> const x;
        std::decay_t<R> r;

        friend void tag_invoke(pika::execution::experimental::start_t,
            operation_state& os) noexcept
        {
            pika::execution::experimental::set_value(
                std::move(os.r), os.x.get());
        };
    };

    template <typename R>
    friend auto tag_invoke(pika::execution::experimental::connect_t,
        const_reference_cuda_sender&& s, R&& r)
    {
        return operation_state<R>{std::move(s.x), std::forward<R>(r)};
    }

    friend cu::cuda_scheduler tag_invoke(
        pika::execution::experimental::get_completion_scheduler_t<
            pika::execution::experimental::set_value_t>,
        const_reference_cuda_sender const& s) noexcept
    {
        return s.sched;
    }
};

struct const_reference_error_cuda_sender
{
    cu::cuda_scheduler sched;

    PIKA_NVCC_PRAGMA_HD_WARNING_DISABLE
    ~const_reference_error_cuda_sender() = default;

    template <template <class...> class Tuple,
        template <class...> class Variant>
    using value_types = Variant<Tuple<>>;

    template <template <class...> class Variant>
    using error_types = Variant<std::exception_ptr const&>;

    static constexpr bool sends_done = false;

    using completion_signatures =
        pika::execution::experimental::completion_signatures<
            pika::execution::experimental::set_value_t(),
            pika::execution::experimental::set_error_t(
                std::exception_ptr const&)>;

    template <typename R>
    struct operation_state
    {
        std::decay_t<R> r;
        friend void tag_invoke(pika::execution::experimental::start_t,
            operation_state& os) noexcept
        {
            auto const e = std::make_exception_ptr(std::runtime_error("error"));
            pika::execution::experimental::set_error(std::move(os.r), e);
        }
    };

    template <typename R>
    friend operation_state<R>
    tag_invoke(pika::execution::experimental::connect_t,
        const_reference_error_cuda_sender, R&& r)
    {
        return {std::forward<R>(r)};
    }

    friend cu::cuda_scheduler tag_invoke(
        pika::execution::experimental::get_completion_scheduler_t<
            pika::execution::experimental::set_value_t>,
        const_reference_error_cuda_sender const& s) noexcept
    {
        return s.sched;
    }
};

struct dummy
{
    static std::atomic<std::size_t> host_void_calls;
    static std::atomic<std::size_t> stream_void_calls;
    static std::atomic<std::size_t> cublas_void_calls;
    static std::atomic<std::size_t> cusolver_void_calls;
    static std::atomic<std::size_t> host_int_calls;
    static std::atomic<std::size_t> stream_int_calls;
    static std::atomic<std::size_t> cublas_int_calls;
    static std::atomic<std::size_t> cusolver_int_calls;
    static std::atomic<std::size_t> host_double_calls;
    static std::atomic<std::size_t> stream_double_calls;
    static std::atomic<std::size_t> cublas_double_calls;
    static std::atomic<std::size_t> cusolver_double_calls;

    static void reset_counts()
    {
        host_void_calls = 0;
        stream_void_calls = 0;
        cublas_void_calls = 0;
        cusolver_void_calls = 0;
        host_int_calls = 0;
        stream_int_calls = 0;
        cublas_int_calls = 0;
        cusolver_int_calls = 0;
        host_double_calls = 0;
        stream_double_calls = 0;
        cublas_double_calls = 0;
        cusolver_double_calls = 0;
    }

    void operator()() const
    {
        ++host_void_calls;
    }

    void operator()(whip::stream_t stream) const
    {
        ++stream_void_calls;
        dummy_kernel<<<1, 1, 0, stream>>>();
    }

    void operator()(hipblasHandle_t) const
    {
        ++cublas_void_calls;
    }

#if !defined(PIKA_HAVE_HIP)
    // hipsolverHandle_t and hipblasHandle_t are the same so this results in a
    // redefinition with HIP.
    void operator()(hipsolverHandle_t) const
    {
        ++cusolver_void_calls;
    }
#endif

    double operator()(int x) const
    {
        ++host_int_calls;
        return x + 1;
    }

    double operator()(int x, whip::stream_t stream) const
    {
        ++stream_int_calls;
        dummy_kernel<<<1, 1, 0, stream>>>();
        return x + 1;
    }

    double operator()(hipblasHandle_t, int x) const
    {
        ++cublas_int_calls;
        return x + 1;
    }

#if !defined(PIKA_HAVE_HIP)
    double operator()(hipsolverHandle_t, int x) const
    {
        ++cusolver_int_calls;
        return x + 1;
    }
#endif

    int operator()(double x) const
    {
        ++host_double_calls;
        return x + 1;
    }

    int operator()(double x, whip::stream_t stream) const
    {
        ++stream_double_calls;
        dummy_kernel<<<1, 1, 0, stream>>>();
        return x + 1;
    }

    int operator()(hipblasHandle_t, double x) const
    {
        ++cublas_double_calls;
        return x + 1;
    }

#if !defined(PIKA_HAVE_HIP)
    int operator()(hipsolverHandle_t, double x) const
    {
        ++cusolver_double_calls;
        return x + 1;
    }
#endif
};

std::atomic<std::size_t> dummy::host_void_calls{0};
std::atomic<std::size_t> dummy::stream_void_calls{0};
std::atomic<std::size_t> dummy::cublas_void_calls{0};
std::atomic<std::size_t> dummy::cusolver_void_calls{0};
std::atomic<std::size_t> dummy::host_int_calls{0};
std::atomic<std::size_t> dummy::stream_int_calls{0};
std::atomic<std::size_t> dummy::cublas_int_calls{0};
std::atomic<std::size_t> dummy::cusolver_int_calls{0};
std::atomic<std::size_t> dummy::host_double_calls{0};
std::atomic<std::size_t> dummy::stream_double_calls{0};
std::atomic<std::size_t> dummy::cublas_double_calls{0};
std::atomic<std::size_t> dummy::cusolver_double_calls{0};

struct dummy_stream
{
    bool& called;
    void operator()(whip::stream_t)
    {
        called = true;
    }
};

struct dummy_cublas
{
    bool& called;
    void operator()(hipblasHandle_t)
    {
        called = true;
    }
};

struct dummy_cusolver
{
    bool& called;
    void operator()(hipsolverHandle_t)
    {
        called = true;
    }
};

__global__ void increment_kernel(int* p)
{
    ++(*p);
}

struct increment
{
    int* operator()(int* p, whip::stream_t stream) const
    {
        increment_kernel<<<1, 1, 0, stream>>>(p);
        return p;
    }
};

auto non_default_constructible_params(
    custom_type_non_default_constructible& x, whip::stream_t)
{
    return std::move(x);
}
auto non_default_constructible_non_copyable_params(
    custom_type_non_default_constructible_non_copyable& x, whip::stream_t)
{
    return std::move(x);
}

int pika_main()
{
    cu::cuda_pool pool{};

    cu::enable_user_polling p;

    // Only stream transform
    {
        dummy::reset_counts();
        auto s = ex::just() | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{});
        // NOTE: then_with_stream calls triggers the receiver on a plain
        // std::thread. We explicitly change the context back to an pika::thread.
        tt::sync_wait(ex::transfer(std::move(s), ex::thread_pool_scheduler{}));
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    {
        dummy::reset_counts();
        auto s = ex::just() | ex::transfer(cu::cuda_scheduler(pool)) |
            cu::then_with_stream(dummy{}) | cu::then_with_stream(dummy{}) |
            cu::then_with_stream(dummy{});
        tt::sync_wait(ex::transfer(std::move(s), ex::thread_pool_scheduler{}));
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(3));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    // Mixing stream transform with host scheduler
    {
        dummy::reset_counts();
        auto s = ex::just() | ex::transfer(cu::cuda_scheduler(pool)) |
            cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{}) |
            ex::transfer(cu::cuda_scheduler(pool)) |
            cu::then_with_stream(dummy{});
        tt::sync_wait(ex::transfer(std::move(s), ex::thread_pool_scheduler{}));
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    {
        dummy::reset_counts();
        auto s = ex::schedule(ex::thread_pool_scheduler{}) | ex::then(dummy{}) |
            ex::transfer(cu::cuda_scheduler(pool)) |
            cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        tt::sync_wait(std::move(s));
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    // Only stream transform with non-void values
    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(
                         std::move(s), ex::thread_pool_scheduler{})),
            2.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) | cu::then_with_stream(dummy{}) |
            cu::then_with_stream(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(
                         std::move(s), ex::thread_pool_scheduler{})),
            4.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
    }

    // Non-copyable or non-default-constructible types
    {
        auto s = ex::just(custom_type_non_default_constructible{42}) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(&non_default_constructible_params);
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(std::move(s),
                                       ex::thread_pool_scheduler{}))
                         .x,
            42);
    }

    {
        auto s =
            ex::just(custom_type_non_default_constructible_non_copyable{42}) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(
                &non_default_constructible_non_copyable_params);
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(std::move(s),
                                       ex::thread_pool_scheduler{}))
                         .x,
            42);
    }

    // Mixing stream transform with host scheduler with non-void values
    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{}) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(ex::transfer(
                         std::move(s), ex::thread_pool_scheduler{})),
            4.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(0));
    }

    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then(dummy{}) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(std::move(s)), 4.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
    }

    {
        dummy::reset_counts();

        auto s = ex::transfer_just(ex::thread_pool_scheduler{}, 1) |
            ex::then(dummy{}) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) | cu::then_with_stream(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(std::move(s)), 5.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
    }

    // Chaining multiple stream transforms without intermediate synchronization
    {
        whip::stream_t first_stream{};
        whip::stream_t second_stream{};
        tt::sync_wait(ex::schedule(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(
                [&](whip::stream_t stream) { first_stream = stream; }) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_EQ(stream, first_stream);
            }) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_EQ(stream, first_stream);
            }) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_NEQ(stream, first_stream);
                second_stream = stream;
            }) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_EQ(stream, second_stream);
            }) |
            cu::then_with_stream([&](whip::stream_t stream) {
                PIKA_TEST_EQ(stream, second_stream);
            }));
    }

    {
        using type = int;
        type p_h = 0;

        type* p;
        whip::malloc(&p, sizeof(type));

        auto s = ex::just(p, &p_h, sizeof(type), whip::memcpy_host_to_device) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(whip::memcpy_async) |
            ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then([p] { return p; }) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(increment{}) |
            cu::then_with_stream(increment{}) |
            cu::then_with_stream(increment{});
        tt::sync_wait(
            ex::when_all(ex::just(&p_h), std::move(s), ex::just(sizeof(type)),
                ex::just(whip::memcpy_device_to_host)) |
            ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(whip::memcpy_async) |
            ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then([&p_h] { PIKA_TEST_EQ(p_h, 3); }) |
            ex::transfer(ex::thread_pool_scheduler{}));

        whip::free(p);
    }

    // cuBLAS and cuSOLVER
    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then(dummy{}) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) |
            cu::then_with_cublas(dummy{}, HIPBLAS_POINTER_MODE_HOST) |
            cu::then_with_cusolver(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(std::move(s)), 6);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cublas_void_calls.load(), std::size_t(0));
#if !defined(PIKA_HAVE_HIP)
        PIKA_TEST_EQ(dummy::cusolver_void_calls.load(), std::size_t(0));
#endif
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cublas_int_calls.load(), std::size_t(1));
#if !defined(PIKA_HAVE_HIP)
        PIKA_TEST_EQ(dummy::cusolver_int_calls.load(), std::size_t(0));
#endif
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
#if defined(PIKA_HAVE_HIP)
        // rocblas_handle and rocsolver_handle being the same, the call to
        // then_with_cusolver results in a increment of the cublas overload.
        PIKA_TEST_EQ(dummy::cublas_double_calls.load(), std::size_t(1));
#else
        PIKA_TEST_EQ(dummy::cublas_double_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cusolver_double_calls.load(), std::size_t(1));
#endif
    }

    {
        dummy::reset_counts();
        auto s = ex::just(1) | ex::transfer(ex::thread_pool_scheduler{}) |
            ex::then(dummy{}) | ex::transfer(cu::cuda_scheduler{pool}) |
            cu::then_with_stream(dummy{}) | cu::then_on_host(dummy{}) |
            cu::then_with_cublas(dummy{}, HIPBLAS_POINTER_MODE_HOST) |
            cu::then_with_cusolver(dummy{}) |
            ex::transfer(ex::thread_pool_scheduler{}) | ex::then(dummy{});
        PIKA_TEST_EQ(tt::sync_wait(std::move(s)), 7.0);
        PIKA_TEST_EQ(dummy::host_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::stream_void_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cublas_void_calls.load(), std::size_t(0));
#if !defined(PIKA_HAVE_HIP)
        PIKA_TEST_EQ(dummy::cusolver_void_calls.load(), std::size_t(0));
#endif
        PIKA_TEST_EQ(dummy::host_int_calls.load(), std::size_t(2));
        PIKA_TEST_EQ(dummy::stream_int_calls.load(), std::size_t(0));
#if defined(PIKA_HAVE_HIP)
        // rocblas_handle and rocsolver_handle being the same, the call to
        // then_with_cusolver results in a increment of the cublas overload.
        PIKA_TEST_EQ(dummy::cublas_int_calls.load(), std::size_t(1));
#else
        PIKA_TEST_EQ(dummy::cublas_int_calls.load(), std::size_t(0));
        PIKA_TEST_EQ(dummy::cusolver_int_calls.load(), std::size_t(1));
#endif
        PIKA_TEST_EQ(dummy::host_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::stream_double_calls.load(), std::size_t(1));
        PIKA_TEST_EQ(dummy::cublas_double_calls.load(), std::size_t(1));
#if !defined(PIKA_HAVE_HIP)
        PIKA_TEST_EQ(dummy::cusolver_double_calls.load(), std::size_t(0));
#endif
    }

    // then_with_stream should be able to handle reference types (by copying
    // them to the operation state)
    {
        int x = 42;
        auto result = tt::sync_wait(cu::then_with_stream(
            const_reference_cuda_sender<int>{x, cu::cuda_scheduler{pool}},
            [](int& x, whip::stream_t) { return x + 1; }));
        PIKA_TEST_EQ(result, 43);
    }

    {
        bool exception_thrown = false;
        try
        {
            tt::sync_wait(cu::then_with_stream(
                const_reference_error_cuda_sender{cu::cuda_scheduler{pool}},
                [](whip::stream_t) { PIKA_TEST(false); }));
        }
        catch (std::runtime_error const& e)
        {
            PIKA_TEST_EQ(std::string(e.what()), std::string("error"));
            exception_thrown = true;
        }
        PIKA_TEST(exception_thrown);
    }

    return pika::finalize();
}

int main(int argc, char* argv[])
{
    PIKA_TEST_EQ_MSG(pika::init(pika_main, argc, argv), 0,
        "pika main exited with non-zero status");

    return 0;
}
