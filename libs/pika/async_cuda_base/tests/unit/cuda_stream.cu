#include "hip/hip_runtime.h"
//  Copyright (c) 2021 ETH Zurich
//
//  SPDX-License-Identifier: BSL-1.0
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <pika/cuda.hpp>
#include <pika/testing.hpp>

#include <whip.hpp>

#include <cstddef>
#include <utility>
#include <vector>

namespace cu = pika::cuda::experimental;

__global__ void kernel(int* p, int i) { p[i] = i * 2; }

int main()
{
    {
        // Default constructed hip_stream uses device 0 and default priority
        cu::hip_stream stream{};

        PIKA_TEST_EQ(stream.get_device(), 0);
        PIKA_TEST_EQ(stream.get_priority(), pika::execution::thread_priority::default_);

        PIKA_TEST_NEQ(stream.get(), whip::stream_t{});

        cu::hip_stream stream2{std::move(stream)};

        PIKA_TEST_EQ(stream.get(), whip::stream_t{});
        PIKA_TEST_NEQ(stream2.get(), whip::stream_t{});
    }

    {
        // We can't really test setting the device properly unless we have
        // multiple devices available, but we test the constructor anyway. The
        // other behaviour should be the same as for a default constructed
        // hip_stream.
        cu::hip_stream stream{0};

        PIKA_TEST_EQ(stream.get_device(), 0);
        PIKA_TEST_EQ(stream.get_priority(), pika::execution::thread_priority::default_);

        PIKA_TEST_NEQ(stream.get(), whip::stream_t{});

        cu::hip_stream stream2{std::move(stream)};

        PIKA_TEST_EQ(stream.get(), whip::stream_t{});
        PIKA_TEST_NEQ(stream2.get(), whip::stream_t{});
    }

    {
        // We should also be able to set the priority.
        cu::hip_stream stream{0, pika::execution::thread_priority::normal};

        PIKA_TEST_EQ(stream.get_device(), 0);
        PIKA_TEST_EQ(stream.get_priority(), pika::execution::thread_priority::normal);

        PIKA_TEST_NEQ(stream.get(), whip::stream_t{});

        cu::hip_stream stream2{std::move(stream)};

        PIKA_TEST_EQ(stream.get(), whip::stream_t{});
        PIKA_TEST_NEQ(stream2.get(), whip::stream_t{});
        PIKA_TEST_EQ(stream.get_priority(), pika::execution::thread_priority::default_);
        PIKA_TEST_EQ(stream2.get_priority(), pika::execution::thread_priority::normal);

        cu::hip_stream stream3{stream};
        cu::hip_stream stream4{stream2};

        PIKA_TEST_EQ(stream3.get(), whip::stream_t{});
        PIKA_TEST_NEQ(stream4.get(), whip::stream_t{});
        PIKA_TEST_NEQ(stream4.get(), stream2.get());
        PIKA_TEST_EQ(stream3.get_priority(), pika::execution::thread_priority::default_);
        PIKA_TEST_EQ(stream4.get_priority(), pika::execution::thread_priority::normal);
    }

    {
        cu::hip_stream stream{0, pika::execution::thread_priority::high};

        PIKA_TEST_EQ(stream.get_device(), 0);
        PIKA_TEST_EQ(stream.get_priority(), pika::execution::thread_priority::high);

        PIKA_TEST_NEQ(stream.get(), whip::stream_t{});

        cu::hip_stream stream2{std::move(stream)};

        PIKA_TEST_EQ(stream.get(), whip::stream_t{});
        PIKA_TEST_NEQ(stream2.get(), whip::stream_t{});
        PIKA_TEST_EQ(stream.get_priority(), pika::execution::thread_priority::default_);
        PIKA_TEST_EQ(stream2.get_priority(), pika::execution::thread_priority::high);

        cu::hip_stream stream3{stream};
        cu::hip_stream stream4{stream2};

        PIKA_TEST_EQ(stream3.get(), whip::stream_t{});
        PIKA_TEST_NEQ(stream4.get(), whip::stream_t{});
        PIKA_TEST_NEQ(stream4.get(), stream2.get());
        PIKA_TEST_EQ(stream3.get_priority(), pika::execution::thread_priority::default_);
        PIKA_TEST_EQ(stream4.get_priority(), pika::execution::thread_priority::high);
    }

    {
        // We should be able to set flags on the stream
        cu::hip_stream stream(0, pika::execution::thread_priority::default_);

        unsigned int expected_flags = 0;
        unsigned int flags = 0;
        whip::stream_get_flags(stream.get(), &flags);
        PIKA_TEST_EQ(stream.get_flags(), expected_flags);
        PIKA_TEST_EQ(flags, expected_flags);

        expected_flags = 0;
        cu::hip_stream stream2{0, pika::execution::thread_priority::default_, expected_flags};

        flags = 0;
        whip::stream_get_flags(stream2.get(), &flags);
        PIKA_TEST_EQ(stream2.get_flags(), expected_flags);
        PIKA_TEST_EQ(flags, expected_flags);

        expected_flags = whip::stream_non_blocking;
        cu::hip_stream stream3{0, pika::execution::thread_priority::default_, expected_flags};

        flags = 0;
        whip::stream_get_flags(stream3.get(), &flags);
        PIKA_TEST_EQ(stream3.get_flags(), expected_flags);
        PIKA_TEST_EQ(flags, expected_flags);
    }

    {
        // Equality is based on the underlying stream.
        cu::hip_stream stream1{0, pika::execution::thread_priority::normal};
        cu::hip_stream stream2{0, pika::execution::thread_priority::normal};
        cu::hip_stream stream3{0, pika::execution::thread_priority::high};

        PIKA_TEST_NEQ(stream1, stream2);
        PIKA_TEST_NEQ(stream1, stream3);

        cu::hip_stream stream4{std::move(stream1)};
        cu::hip_stream stream5{std::move(stream2)};
        cu::hip_stream stream6{std::move(stream3)};

        PIKA_TEST_EQ(stream1, stream2);
        PIKA_TEST_EQ(stream1, stream3);
        PIKA_TEST_NEQ(stream4, stream5);
        PIKA_TEST_NEQ(stream4, stream6);
    }

    {
        // We can schedule work with the underlying stream in a hip_stream.
        std::vector<cu::hip_stream> streams;
        streams.emplace_back();
        streams.emplace_back();
        streams.emplace_back(0);
        streams.emplace_back(0, pika::execution::thread_priority::normal);
        streams.emplace_back(0, pika::execution::thread_priority::high);
        // The first stream should stay usable after this
        streams.push_back(std::move(streams[0]));

        int* p;
        whip::malloc(&p, sizeof(int) * streams.size());

        for (std::size_t i = 0; i < streams.size(); ++i)
        {
            kernel<<<1, 1, 0, streams[i].get()>>>(p, i);
            whip::check_last_error();
        }

        whip::device_synchronize();
        std::vector<int> s(streams.size(), 0);

        whip::memcpy(s.data(), p, sizeof(int) * streams.size(), whip::memcpy_device_to_host);
        whip::free(p);

        for (int i = 0; i < static_cast<int>(streams.size()); ++i) { PIKA_TEST_EQ(s[i], i * 2); }
    }
}
