#include "hip/hip_runtime.h"
//  Copyright (c) 2021 ETH Zurich
//
//  SPDX-License-Identifier: BSL-1.0
//  Distributed under the Boost Software License, Version 1.0. (See accompanying
//  file LICENSE_1_0.txt or copy at http://www.boost.org/LICENSE_1_0.txt)

#include <pika/cuda.hpp>
#include <pika/testing.hpp>

#include <whip.hpp>

#include <utility>

#if defined(PIKA_HAVE_HIP)
# define hipblasSasum rocblas_sasum
# define HIPBLAS_STATUS_NOT_INITIALIZED rocblas_status_invalid_handle
#endif

namespace cu = pika::cuda::experimental;

__global__ void kernel(float* p)
{
    int const i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i] = i;
}

int main()
{
    cu::hip_stream stream;

    {
        // Default constructed cublas_handle uses device 0 and default priority
        cu::cublas_handle handle{};

        PIKA_TEST_EQ(handle.get_device(), 0);
        PIKA_TEST_EQ(handle.get_stream(), whip::stream_t{0});

        PIKA_TEST_NEQ(handle.get(), hipblasHandle_t{});

        cu::cublas_handle handle2{std::move(handle)};

        PIKA_TEST_EQ(handle.get(), hipblasHandle_t{});
        PIKA_TEST_NEQ(handle2.get(), hipblasHandle_t{});

        cu::cublas_handle handle3{handle};
        cu::cublas_handle handle4{handle2};

        PIKA_TEST_EQ(handle3.get(), hipblasHandle_t{});
        PIKA_TEST_NEQ(handle4.get(), hipblasHandle_t{});
        PIKA_TEST_NEQ(handle4.get(), handle2.get());
    }

    {
        // Equality is based on the underlying handle.
        cu::cublas_handle handle1{stream};
        cu::cublas_handle handle2{stream};
        cu::cublas_handle handle3{stream};

        PIKA_TEST_NEQ(handle1.get_stream(), whip::stream_t{0});
        PIKA_TEST_NEQ(handle2.get_stream(), whip::stream_t{0});
        PIKA_TEST_NEQ(handle3.get_stream(), whip::stream_t{0});
        PIKA_TEST_EQ(handle1.get_stream(), stream.get());
        PIKA_TEST_EQ(handle2.get_stream(), stream.get());
        PIKA_TEST_EQ(handle3.get_stream(), stream.get());
        PIKA_TEST_NEQ(handle1, handle2);
        PIKA_TEST_NEQ(handle1, handle3);

        cu::cublas_handle handle4{std::move(handle1)};
        cu::cublas_handle handle5{std::move(handle2)};
        cu::cublas_handle handle6{std::move(handle3)};

        PIKA_TEST_EQ(handle1, handle2);
        PIKA_TEST_EQ(handle1, handle3);
        PIKA_TEST_NEQ(handle4, handle5);
        PIKA_TEST_NEQ(handle4, handle6);
    }

    {
        // Equality is based on the underlying handle.
        cu::cublas_handle handle1{};
        cu::cublas_handle handle2{};
        cu::cublas_handle handle3{};

        handle1.set_stream(stream);
        handle2.set_stream(stream);
        handle3.set_stream(stream);

        PIKA_TEST_NEQ(handle1.get_stream(), whip::stream_t{0});
        PIKA_TEST_NEQ(handle2.get_stream(), whip::stream_t{0});
        PIKA_TEST_NEQ(handle3.get_stream(), whip::stream_t{0});
        PIKA_TEST_EQ(handle1.get_stream(), stream.get());
        PIKA_TEST_EQ(handle2.get_stream(), stream.get());
        PIKA_TEST_EQ(handle3.get_stream(), stream.get());
        PIKA_TEST_NEQ(handle1, handle2);
        PIKA_TEST_NEQ(handle1, handle3);

        cu::cublas_handle handle4{std::move(handle1)};
        cu::cublas_handle handle5{std::move(handle2)};
        cu::cublas_handle handle6{std::move(handle3)};

        PIKA_TEST_EQ(handle1, handle2);
        PIKA_TEST_EQ(handle1, handle3);
        PIKA_TEST_NEQ(handle4, handle5);
        PIKA_TEST_NEQ(handle4, handle6);
    }

    {
        // We can schedule work with the underlying handle in a cublas_handle.
        cu::cublas_handle handle{stream};

        int const n = 100;
        float* p;
        whip::malloc(&p, sizeof(float) * n);

        kernel<<<n, 1, 0, handle.get_stream()>>>(p);
        whip::check_last_error();
        whip::device_synchronize();
        float r;
        handle.set_pointer_mode(HIPBLAS_POINTER_MODE_HOST);
        cu::check_cublas_error(hipblasSasum(handle.get(), n, p, 1, &r));
        whip::device_synchronize();

        whip::free(p);

        PIKA_TEST_EQ(r, (n * (n - 1) / 2));
    }

    {
        // A moved-from handle is invalid and will give an error if used
        cu::cublas_handle handle{stream};
        cu::cublas_handle handle2{std::move(handle)};

        try
        {
            cu::check_cublas_error(hipblasSetPointerMode(handle.get(), HIPBLAS_POINTER_MODE_HOST));
            PIKA_TEST(false);
        }
        catch (cu::cublas_exception const& e)
        {
            PIKA_TEST_EQ(static_cast<int>(e.get_cublas_errorcode()),
                static_cast<int>(HIPBLAS_STATUS_NOT_INITIALIZED));
        }
        catch (...)
        {
            PIKA_TEST(false);
        }
    }
}
